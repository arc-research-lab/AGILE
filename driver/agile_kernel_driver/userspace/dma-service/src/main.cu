#include "hip/hip_runtime.h"
#include <iostream>
#include <thread>
#include <chrono>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include <sys/syscall.h>
#include <unistd.h>

#include <boost/program_options.hpp>
#include <fstream>

#include "agile_driver.h"
#include "t_agile_service.h"
#include "agile_helper.h"

#include "config.h"

#include "io_utils.h"

__global__ 
void dma_issue_kernel(AgileDmaEngine * engine, size_t size, uint32_t *src_idx, uint32_t *dst_idx, char flags, uint32_t total_cmd, uint32_t repeat){

    uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    AgileLockChain chain;
    uint32_t *lock;
    for(uint32_t rep = 0; rep < repeat; ++rep){
        for(uint32_t i = tid; i < total_cmd; i += blockDim.x * gridDim.x){
            // Submit DMA request
            engine->submit(i * size, i * size, size, flags, &lock, &chain);
            wait_cmd(lock);
        }
    }
}

int main(int argc, char ** argv){

    Configs conf(argc, argv);
    conf.parse();

    // Initialize CUDA
    ASSERTDRV(hipInit(0));
    hipDevice_t dev;
    ASSERTDRV(hipDeviceGet(&dev, 0));

    hipCtx_t dev_ctx;
    ASSERTDRV(hipDevicePrimaryCtxRetain(&dev_ctx, dev));
    ASSERTDRV(hipCtxSetCurrent(dev_ctx));

    INFO("main starting...");

    AgileDriver driver; // can only have one instance.

    INFO("Total", driver.getAvaiableCpuDma(), "DMA engines available.");
    driver.setDmaQueuePairCudaDma(0, 128);
    driver.setDmaQueuePairCpuDma(1, 128);
    driver.setMonitorThreadsNum(1);
    driver.setWorkerThreadsNum(1);
    driver.setHbmCacheSize(1024l * 1024l * 1024l); // 1GB HBM cache
    driver.setDramCacheSize(1024l * 1024l * 1024l); // TODO: this is reserved in grub.

    driver.allocateHost();

    
    
    driver.startMonitors();
    driver.startWorkers();

    int numBlocksPerSM = 0;
    cuda_err_chk(hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &numBlocksPerSM,
        pollingService,
        1024, // threads per bc       
        0
    ));

    cuda_err_chk(hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &numBlocksPerSM,
        dma_issue_kernel,
        1024, // threads per bc       
        0
    ));

    hipStream_t kernel_s;
    hipStream_t service_s;
    cuda_err_chk(hipStreamCreateWithFlags(&kernel_s, hipStreamNonBlocking));
    cuda_err_chk(hipStreamCreateWithFlags(&service_s, hipStreamNonBlocking));

    AgileGpuMem reserved_hbm_mem = driver.getReservedMem();
    pollingService<<<1, 1024, 0, service_s>>>(driver.getAgileDmaQueuePairDevicePtr(), driver.getDmaQueuePairNum(), (uint32_t *)reserved_hbm_mem.d_ptr);
    auto start = std::chrono::high_resolution_clock::now();
    dma_issue_kernel<<<1, 32, 0, kernel_s>>>(driver.getDmaEngineDevice(), conf.transfer_size, nullptr, nullptr, DMA_CPU2GPU, conf.command_num, conf.repeat);
    cuda_err_chk(hipStreamSynchronize(kernel_s));
    auto end = std::chrono::high_resolution_clock::now();

    stop_service((uint32_t *)reserved_hbm_mem.h_ptr);
    cuda_err_chk(hipStreamSynchronize(service_s));

    double total_time = std::chrono::duration<double, std::milli>(end - start).count();
    double total_gb = ((double)conf.transfer_size * conf.command_num * conf.repeat) / (1024.0 * 1024.0 * 1024.0);
    INFO("Total data:", total_gb, "GB");
    INFO("Total time:", total_time, "ms");
    INFO("Throughput:", total_gb / (total_time / 1000.0), "GB/s");

    cuda_err_chk(hipStreamDestroy(kernel_s));
    cuda_err_chk(hipStreamDestroy(service_s));


    driver.stopMonitors();
    driver.stopWorkers();
    driver.freeHost();

    return 0;
}