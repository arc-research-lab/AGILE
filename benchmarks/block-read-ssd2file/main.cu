#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <cstdio>

#include "agile_host.h"
#include "config.h"
#include "../common/cache_impl.h"
#include "../common/table_impl.h"

#define CPU_CACHE_IMPL DisableCPUCache
#define SHARE_TABLE_IMPL DisableShareTable
#define GPU_CACHE_IMPL SimpleGPUCache<CPU_CACHE_IMPL, SHARE_TABLE_IMPL>

#define AGILE_CTRL AgileCtrl<GPU_CACHE_IMPL, CPU_CACHE_IMPL, SHARE_TABLE_IMPL>
#define AGILE_HOST AgileHost<GPU_CACHE_IMPL, CPU_CACHE_IMPL, SHARE_TABLE_IMPL>

__global__ void read_kernel(AGILE_CTRL * ctrl, AgileBuf * buf, unsigned int block_offset){
    AgileLockChain chain;
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    AgileBufPtr bufPtr(buf[tid]);
    bufPtr.resetStatus();
    ctrl->asyncRead(0, block_offset + tid, bufPtr, chain);
    bufPtr.wait();
}

int main(int argc, char ** argv){
    Configs cfg(argc, argv);

    AGILE_HOST host(0, cfg.slot_size);    

    CPU_CACHE_IMPL c_cache(0, cfg.slot_size); // Disable CPU cache
    SHARE_TABLE_IMPL w_table(0); // Disable write table
    GPU_CACHE_IMPL g_cache(cfg.gpu_slot_num, cfg.slot_size);

    host.setGPUCache(g_cache);
    host.setCPUCache(c_cache);
    host.setShareTable(w_table);

    host.addNvmeDev(cfg.nvme_bar, cfg.bar_size, cfg.ssd_blk_offset, cfg.queue_num, cfg.queue_depth);
    host.initNvme();

    int numBlocksPerSM1 = 0;
    hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &numBlocksPerSM1,
        start_agile_cq_service<GPU_CACHE_IMPL, CPU_CACHE_IMPL, SHARE_TABLE_IMPL>,
        1024, // threads per block
        0    // dynamic shared memory
    );

    int numBlocksPerSM2 = 0;
    hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &numBlocksPerSM2,
        read_kernel,
        cfg.thread_dim, // threads per block
        0    // dynamic shared memory
    );

    std::cout << "numBlocksPerSM1: " << numBlocksPerSM1 << std::endl;
    std::cout << "numBlocksPerSM2: " << numBlocksPerSM2 << std::endl;

    AgileBuf * buf;
    host.allocateBuffer(buf, cfg.block_dim * cfg.thread_dim);
    host.configParallelism(cfg.block_dim, cfg.thread_dim, cfg.agile_dim);
    host.initializeAgile();
    remove(cfg.output_file.c_str());
    auto *ctrl = host.getAgileCtrlDevicePtr();
    host.startAgile();
    for(unsigned long i = 0; i < cfg.ssd_blk_num; i += cfg.block_dim * cfg.thread_dim){
        std::cout << "\033[F" << "progress: " << i <<  " / " << cfg.ssd_blk_num << "    " << std::endl;
        host.runKernel(read_kernel, ctrl, buf, i);
        host.appendBuf2File(cfg.output_file, buf, cfg.block_dim * cfg.thread_dim);
    }
    host.stopAgile();
    host.freeBuffer(buf, cfg.block_dim * cfg.thread_dim);
    host.closeNvme();

    return 0;
}